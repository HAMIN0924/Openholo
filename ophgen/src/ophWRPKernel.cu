#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install, copy or use the software.
//
//
//                           License Agreement
//                For Open Source Digital Holographic Library
//
// Openholo library is free software;
// you can redistribute it and/or modify it under the terms of the BSD 2-Clause license.
//
// Copyright (C) 2017-2024, Korea Electronics Technology Institute. All rights reserved.
// E-mail : contact.openholo@gmail.com
// Web : http://www.openholo.org
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//  1. Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//  2. Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the copyright holder or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
// This software contains opensource software released under GNU Generic Public License,
// NVDIA Software License Agreement, or CUDA supplement to Software License Agreement.
// Check whether software you use contains licensed software.
//
//M*/

#ifndef ophWRPKernel_cu__
#define ophWRPKernel_cu__

#include "ophKernel.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand_uniform.h>
#include <>
#include "ophWRP_GPU.h"

__global__ void cudaKernel_CalcData(hipfftDoubleComplex *src, const WRPGpuConst* config)
{
	ulonglong tid = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ double ppX;
	__shared__ double ppY;
	__shared__ int pnX;
	__shared__ int pnY;
	__shared__ int pnXY;
	__shared__ double ssX;
	__shared__ double ssY;
	__shared__ double z;
	__shared__ double v;
	__shared__ double lambda;
	__shared__ double distance;
	__shared__ double pi2;

	if (threadIdx.x == 0)
	{
		ppX = config->pp_X;
		ppY = config->pp_Y;
		pnX = config->pn_X;
		pnY = config->pn_Y;
		pnXY = pnX * pnY;
		ssX = pnX * ppX * 2;
		ssY = pnY * ppY * 2;
		lambda = config->lambda;
		distance = config->propa_d;
		pi2 = config->pi2;
		z = distance * pi2;
		v = 1 / (lambda * lambda);
	}
	__syncthreads();

	if (tid < pnXY * 4)
	{
		int pnX2 = pnX * 2;

		int w = tid % pnX2;
		int h = tid / pnX2;

		double fy = (-pnY + h) / ssY;
		double fyy = fy * fy;
		double fx = (-pnX + w) / ssX;
		double fxx = fx * fx;
		double sqrtpart = sqrt(v - fxx - fyy);

		hipDoubleComplex prop;
		prop.x = 0;
		prop.y = z * sqrtpart;

		exponent_complex(&prop);

		hipDoubleComplex val;
		val.x = src[tid].x;
		val.y = src[tid].y;

		hipDoubleComplex val2 = hipCmul(val, prop);
		src[tid].x = val2.x;
		src[tid].y = val2.y;
	}
}

__global__ void cudaKernel_MoveDataPost(hipDoubleComplex *src, hipDoubleComplex *dst, const WRPGpuConst* config)
{
	ulonglong tid = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ int pnX;
	__shared__ int pnY;
	__shared__ ulonglong pnXY;

	if (threadIdx.x == 0)
	{
		pnX = config->pn_X;
		pnY = config->pn_Y;
		pnXY = pnX * pnY;
	}
	__syncthreads();

	if (tid < pnXY)
	{
		int w = tid % pnX;
		int h = tid / pnX;
		ulonglong iSrc = pnX * 2 * (pnY / 2 + h) + pnX / 2;

		dst[tid] = src[iSrc + w];
	}
}

__global__ void cudaKernel_MoveDataPre(hipDoubleComplex *src, hipDoubleComplex *dst, const WRPGpuConst* config)
{
	ulonglong tid = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ int pnX;
	__shared__ int pnY;
	__shared__ ulonglong pnXY;

	if (threadIdx.x == 0)
	{
		pnX = config->pn_X;
		pnY = config->pn_Y;
		pnXY = pnX * pnY;
	}
	__syncthreads();

	if (tid < pnXY)
	{
		int w = tid % pnX;
		int h = tid / pnX;
		ulonglong iDst = pnX * 2 * (pnY / 2 + h) + pnX / 2;
		dst[iDst + w] = src[tid];
	}
}

__global__ void cudaKernel_GenWRP(Real* pc_dst, Real* amp_dst, const WRPGpuConst* config, const int n_points_stream, hipDoubleComplex* dst)
{
	ulonglong tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < n_points_stream)
	{
		__shared__ double ppX;
		__shared__ double ppY;
		__shared__ int pnX;
		__shared__ int pnY;
		__shared__ double dz;
		__shared__ double dzz;
		__shared__ double pi2;
		__shared__ double k;
		__shared__ double lambda;
		__shared__ bool random_phase;
		__shared__ bool sign;

		if (threadIdx.x == 0) {
			ppX = config->pp_X;
			ppY = config->pp_Y;
			pnX = config->pn_X;
			pnY = config->pn_Y;
			dz = config->wrp_d - config->zmax;
			dzz = dz * dz;
			k = config->k;
			lambda = config->lambda;
			pi2 = config->pi2;
			random_phase = config->bRandomPhase;
			sign = dz > 0.0 ? true : false;
		}
		__syncthreads();

		int idx = tid * 3;
		double x = pc_dst[idx + _X];
		double y = pc_dst[idx + _Y];
		double z = pc_dst[idx + _Z];
		double amp = amp_dst[idx + config->iAmplitude];

		int hpnX = pnX / 2;
		int hpnY = pnY / 2;
		double ppXX = ppX * ppX * 2;
		//double dz = config->wrp_d - config->zmax;
		double tw = fabs(lambda * dz / ppXX) * 2;

		int w = (int)tw;
		int tx = (int)(x / ppX) + hpnX;
		int ty = (int)(y / ppY) + hpnY;

		hiprandState state;
		if (random_phase)
		{
			hiprand_init(4 * w * w, 0, 0, &state);
		}

		for (int wy = -w; wy < w; wy++)
		{
			double dy = wy * ppY;
			double dyy = dy * dy;
			int tmpY = wy + ty;
			int baseY = tmpY * pnX;

			for (int wx = -w; wx < w; wx++) //WRP coordinate
			{
				int tmpX = wx + tx;

				if (tmpX >= 0 && tmpX < pnX && tmpY >= 0 && tmpY < pnY) {
					int iDst = tmpX + baseY;

					double dx = wx * ppX;

					double r = sign ? sqrt(dx * dx + dyy + dzz) : -sqrt(dx * dx + dyy + dzz);
					double randomData = random_phase ? hiprand_uniform_double(&state) : 1.0;
					double randVal = randomData * pi2;

					hipDoubleComplex tmp;
					tmp.x = (amp * cos(k*r) * cos(randVal)) / r;
					tmp.y = (-amp * sin(k*r) * sin(randVal)) / r;

#if defined(__cplusplus) && defined(__HIPCC__)
#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
					dst[iDst].x = atomicAdd(&dst[iDst].x, tmp.x);
					dst[iDst].y = atomicAdd(&dst[iDst].y, tmp.y);

#else
					dst[iDst].x += tmp.x; // <-- sync problem
					dst[iDst].y += tmp.y; // <-- sync problem
#endif					
#else
					dst[iDst].x += tmp.x; // <-- sync problem
					dst[iDst].y += tmp.y; // <-- sync problem
#endif
				}
			}
		}
	}
}

extern "C"
{
	void cudaFresnelPropagationWRP(
		const int &nBlocks, const int&nBlocks2, const int &nThreads, const int &nx, const int &ny,
		hipDoubleComplex *src, hipDoubleComplex *dst, hipfftDoubleComplex *fftsrc, hipfftDoubleComplex *fftdst,
		const WRPGpuConst* cuda_config)
	{
		cudaKernel_MoveDataPre << <nBlocks, nThreads >> > (src, dst, cuda_config);

		cudaFFT(nullptr, nx * 2, ny * 2, dst, fftsrc, HIPFFT_FORWARD, false);

		cudaKernel_CalcData << <nBlocks2, nThreads >> > (fftsrc, cuda_config);

		cudaFFT(nullptr, nx * 2, ny * 2, fftsrc, fftdst, HIPFFT_BACKWARD, true);

		cudaKernel_MoveDataPost << <nBlocks, nThreads >> > (fftdst, src, cuda_config);
	}

	void cudaGenWRP(
		const int &nBlocks, const int &nThreads, const int &n_pts_per_stream,
		Real* cuda_pc_data, Real* cuda_amp_data,
		hipDoubleComplex* cuda_dst, const WRPGpuConst* cuda_config)
	{
		cudaKernel_GenWRP << <nBlocks, nThreads >> > (cuda_pc_data, cuda_amp_data, cuda_config, n_pts_per_stream, cuda_dst);
	}
}

#endif // !OphWRPKernel_cu__