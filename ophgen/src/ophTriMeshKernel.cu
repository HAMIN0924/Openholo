#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install, copy or use the software.
//
//
//                           License Agreement
//                For Open Source Digital Holographic Library
//
// Openholo library is free software;
// you can redistribute it and/or modify it under the terms of the BSD 2-Clause license.
//
// Copyright (C) 2017-2024, Korea Electronics Technology Institute. All rights reserved.
// E-mail : contact.openholo@gmail.com
// Web : http://www.openholo.org
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//  1. Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//  2. Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the copyright holder or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
// This software contains opensource software released under GNU Generic Public License,
// NVDIA Software License Agreement, or CUDA supplement to Software License Agreement.
// Check whether software you use contains licensed software.
//
//M*/
#pragma once
#ifndef ophTriMeshKernel_cu__
#define ophTriMeshKernel_cu__
#include "ophKernel.cuh"
#include "ophTriMesh_GPU.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>
#include <vector>

__device__  void exponent_complex_mesh(hipDoubleComplex* val)
{
	double exp_val = exp(val->x);
	double cos_v;
	double sin_v;
	sincos(val->y, &sin_v, &cos_v);

	val->x = exp_val * cos_v;
	val->y = exp_val * sin_v;
}

__device__  void exponent_complex_meshf(hipFloatComplex* val)
{
	float exp_val = expf(val->x);
	float cos_v;
	float sin_v;
	sincosf(val->y, &sin_v, &cos_v);

	val->x = exp_val * cos_v;
	val->y = exp_val * sin_v;
}


void cudaFFT_Mesh(ihipStream_t* stream, int nx, int ny, hipfftDoubleComplex* in_field, hipfftDoubleComplex* output_field, int direction)
{
	unsigned int nblocks = (nx * ny + kBlockThreads - 1) / kBlockThreads;
	int N = nx * ny;
	fftShift << <nblocks, kBlockThreads, 0, stream >> > (N, nx, ny, in_field, output_field, false);

	hipfftHandle plan;

	// fft
	if (hipfftPlan2d(&plan, ny, nx, HIPFFT_Z2Z) != HIPFFT_SUCCESS)
	{
		//LOG("FAIL in creating cufft plan");
		return;
	};

	hipfftResult result;

	if (direction == -1)
		result = hipfftExecZ2Z(plan, output_field, in_field, HIPFFT_FORWARD);
	else
		result = hipfftExecZ2Z(plan, output_field, in_field, HIPFFT_BACKWARD);

	if (result != HIPFFT_SUCCESS)
	{
		//LOG("------------------FAIL: execute cufft, code=%s", result);
		return;
	}

	if (hipDeviceSynchronize() != hipSuccess) {
		//LOG("Cuda error: Failed to synchronize\n");
		return;
	}

	fftShift << < nblocks, kBlockThreads, 0, stream >> > (N, nx, ny, in_field, output_field, false);

	hipfftDestroy(plan);
}

void cudaFFT_Meshf(ihipStream_t* stream, int nx, int ny, hipfftComplex* in_field, hipfftComplex* output_field, int direction)
{
	unsigned int nblocks = (nx * ny + kBlockThreads - 1) / kBlockThreads;
	int N = nx * ny;
	fftShiftf << <nblocks, kBlockThreads, 0, stream >> > (N, nx, ny, in_field, output_field, false);

	hipfftHandle plan;

	// fft
	if (hipfftPlan2d(&plan, ny, nx, HIPFFT_C2C) != HIPFFT_SUCCESS)
	{
		//LOG("FAIL in creating cufft plan");
		return;
	};

	hipfftResult result;

	if (direction == -1)
		result = hipfftExecC2C(plan, output_field, in_field, HIPFFT_FORWARD);
	else
		result = hipfftExecC2C(plan, output_field, in_field, HIPFFT_BACKWARD);

	if (result != HIPFFT_SUCCESS)
	{
		//LOG("------------------FAIL: execute cufft, code=%s", result);
		return;
	}

	if (hipDeviceSynchronize() != hipSuccess) {
		//LOG("Cuda error: Failed to synchronize\n");
		return;
	}

	fftShiftf << < nblocks, kBlockThreads, 0, stream >> > (N, nx, ny, in_field, output_field, false);

	hipfftDestroy(plan);
}

__global__
void cudaKernel_double_RefAS_flat(hipfftDoubleComplex* output, const MeshKernelConfig* config,
	double shadingFactor, const geometric* geom, double carrierWaveX, double carrierWaveY, double carrierWaveZ)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if (tid < config->pn_X * config->pn_Y) {

		int col = tid % config->pn_X;
		int row = tid / config->pn_X;

		double flx, fly, flz, fx, fy, fz, flxShifted, flyShifted, freqTermX, freqTermY;

		double det = geom->loRot[0] * geom->loRot[3] - geom->loRot[1] * geom->loRot[2];
		if (det == 0)
			return;

		double a = 1 / det;
		double invLoRot[4];
		invLoRot[0] = a * geom->loRot[3];
		invLoRot[1] = -a * geom->loRot[2];
		invLoRot[2] = -a * geom->loRot[1];
		invLoRot[3] = a * geom->loRot[0];

		hipDoubleComplex refTerm1 = make_hipDoubleComplex(0, 0);
		hipDoubleComplex refTerm2 = make_hipDoubleComplex(0, 0);
		hipDoubleComplex refTerm3 = make_hipDoubleComplex(0, 0);
		hipDoubleComplex refAS = make_hipDoubleComplex(0, 0);
		hipDoubleComplex term1 = make_hipDoubleComplex(0, 0);
		hipDoubleComplex term2 = make_hipDoubleComplex(0, 0);

		term1.y = -config->pi2 / config->lambda * (
			carrierWaveX * (geom->glRot[0] * geom->glShift[0] + geom->glRot[3] * geom->glShift[1] + geom->glRot[6] * geom->glShift[2])
			+ carrierWaveY * (geom->glRot[1] * geom->glShift[0] + geom->glRot[4] * geom->glShift[1] + geom->glRot[7] * geom->glShift[2])
			+ carrierWaveZ * (geom->glRot[2] * geom->glShift[0] + geom->glRot[5] * geom->glShift[1] + geom->glRot[8] * geom->glShift[2]));


		// calculate frequency term =======================================================================
		int idxFx = -config->pn_X / 2 + col;
		int idxFy = config->pn_X / 2 - row;
		double w = 1.0 / config->lambda;

		fx = (double)idxFx * config->dfx;
		fy = (double)idxFy * config->dfy;
		fz = sqrt(w * w - fx * fx - fy * fy);

		flx = geom->glRot[0] * fx + geom->glRot[1] * fy + geom->glRot[2] * fz;
		fly = geom->glRot[3] * fx + geom->glRot[4] * fy + geom->glRot[5] * fz;
		flz = sqrt(w * w - flx * flx - fly * fly);


		flxShifted = flx - w * (geom->glRot[0] * carrierWaveX + geom->glRot[1] * carrierWaveY + geom->glRot[2] * carrierWaveZ);
		flyShifted = fly - w * (geom->glRot[3] * carrierWaveX + geom->glRot[4] * carrierWaveY + geom->glRot[5] * carrierWaveZ);
		freqTermX = invLoRot[0] * flxShifted + invLoRot[1] * flyShifted;
		freqTermY = invLoRot[2] * flxShifted + invLoRot[3] * flyShifted;

		double sqFreqTermX = freqTermX * freqTermX;
		double cuFreqTermX = sqFreqTermX * freqTermX;
		double sqFreqTermY = freqTermY * freqTermY;
		double cuFreqTermY = sqFreqTermY * freqTermY;

		//if (freqTermX == -freqTermY && freqTermY != 0) {
		if (abs(freqTermX - freqTermY) <= config->tolerence && abs(freqTermY) > config->tolerence) {
			refTerm1.y = config->pi2 * freqTermY;
			refTerm2.y = 1;

			//refAS = shadingFactor * (((Complex<Real>)1 - exp(refTerm1)) / (4 * pi*pi*freqTermY * freqTermY) + refTerm2 / (2 * pi*freqTermY));
			exponent_complex_mesh(&refTerm1);
			hipDoubleComplex value1 = make_hipDoubleComplex(1, 0);
			hipDoubleComplex value2 = hipCsub(value1, refTerm1);
			double value3 = config->square_pi2 * freqTermY * freqTermY;
			hipDoubleComplex value4 = hipCdiv(value2, make_hipDoubleComplex(value3, 0));
			hipDoubleComplex value5 = hipCdiv(refTerm2, make_hipDoubleComplex(config->pi2 * freqTermY, 0));
			hipDoubleComplex value6 = hipCadd(value4, value5);
			refAS = hipCmul(value6, make_hipDoubleComplex(shadingFactor, 0));

			//}else if (freqTermX == freqTermY && freqTermX == 0) {
		}
		else if (abs(freqTermX - freqTermY) <= config->tolerence && abs(freqTermX) <= config->tolerence) {

			//refAS = shadingFactor * 1 / 2;
			refAS = make_hipDoubleComplex(shadingFactor * 0.5, 0);

			//} else if (freqTermX != 0 && freqTermY == 0) {
		}
		else if (abs(freqTermX) > config->tolerence && abs(freqTermY) <= config->tolerence) {

			refTerm1.y = -config->pi2 * freqTermX;
			refTerm2.y = 1;

			//refAS = shadingFactor * ((exp(refTerm1) - (Complex<Real>)1) / (2 * M_PI*freqTermX * 2 * M_PI*freqTermX) + (refTerm2 * exp(refTerm1)) / (2 * M_PI*freqTermX));
			exponent_complex_mesh(&refTerm1);
			hipDoubleComplex value1 = make_hipDoubleComplex(1, 0);
			hipDoubleComplex value2 = hipCsub(refTerm1, value1);
			double value3 = config->square_pi2 * sqFreqTermX;
			hipDoubleComplex value4 = hipCdiv(value2, make_hipDoubleComplex(value3, 0));

			hipDoubleComplex value5 = hipCmul(refTerm2, refTerm1);
			hipDoubleComplex value6 = hipCdiv(value5, make_hipDoubleComplex(config->pi2 * freqTermX, 0));

			hipDoubleComplex value7 = hipCadd(value4, value6);
			refAS = hipCmul(value7, make_hipDoubleComplex(shadingFactor, 0));

			//} else if (freqTermX == 0 && freqTermY != 0) {
		}
		else if (abs(freqTermX) <= config->tolerence && abs(freqTermY) > config->tolerence) {

			refTerm1.y = config->pi2 * freqTermY;
			refTerm2.y = 1;

			//refAS = shadingFactor * (((Complex<Real>)1 - exp(refTerm1)) / (4 * M_PI*M_PI*freqTermY * freqTermY) - refTerm2 / (2 * M_PI*freqTermY));
			exponent_complex_mesh(&refTerm1);
			hipDoubleComplex value1 = make_hipDoubleComplex(1, 0);
			hipDoubleComplex value2 = hipCsub(value1, refTerm1);
			double value3 = config->square_pi2 * sqFreqTermY;
			hipDoubleComplex value4 = hipCdiv(value2, make_hipDoubleComplex(value3, 0));
			hipDoubleComplex value5 = hipCdiv(refTerm2, make_hipDoubleComplex(config->pi2 * freqTermY, 0));
			hipDoubleComplex value6 = hipCsub(value4, value5);
			refAS = hipCmul(value6, make_hipDoubleComplex(shadingFactor, 0));

		}
		else {

			refTerm1.y = -config->pi2 * freqTermX;
			refTerm2.y = -config->pi2 * (freqTermX + freqTermY);

			//refAS = shadingFactor * ((exp(refTerm1) - (Complex<Real>)1) / (4 * M_PI*M_PI*freqTermX * freqTermY) + ((Complex<Real>)1 - exp(refTerm2)) / (4 * M_PI*M_PI*freqTermY * (freqTermX + freqTermY)));
			exponent_complex_mesh(&refTerm1);
			hipDoubleComplex value1 = make_hipDoubleComplex(1, 0);
			hipDoubleComplex value2 = hipCsub(refTerm1, value1);
			double value3 = config->square_pi2 * freqTermX * freqTermY;
			hipDoubleComplex value4 = hipCdiv(value2, make_hipDoubleComplex(value3, 0));

			exponent_complex_mesh(&refTerm2);
			hipDoubleComplex value5 = hipCsub(make_hipDoubleComplex(1, 0), refTerm2);
			double value6 = config->square_pi2 * freqTermY * (freqTermX + freqTermY);
			hipDoubleComplex value7 = hipCdiv(value5, make_hipDoubleComplex(value6, 0));

			hipDoubleComplex value8 = hipCadd(value4, value7);
			refAS = hipCmul(value8, make_hipDoubleComplex(shadingFactor, 0));
		}

		hipDoubleComplex temp;
		if (abs(fz) <= config->tolerence)
			temp = make_hipDoubleComplex(0, 0);
		else {
			term2.y = config->pi2 * (flx * geom->glShift[0] + fly * geom->glShift[1] + flz * geom->glShift[2]);

			//temp = refAS / det * exp(term1)* flz / fz * exp(term2);

			exponent_complex_mesh(&term1);
			exponent_complex_mesh(&term2);

			hipDoubleComplex tmp1 = hipCdiv(refAS, make_hipDoubleComplex(det, 0));
			hipDoubleComplex tmp2 = hipCmul(tmp1, term1);
			hipDoubleComplex tmp3 = hipCmul(tmp2, make_hipDoubleComplex(flz, 0));
			hipDoubleComplex tmp4 = hipCdiv(tmp3, make_hipDoubleComplex(fz, 0));
			temp = hipCmul(tmp4, term2);

		}

		double absval = sqrt((temp.x * temp.x) + (temp.y * temp.y));
		if (absval > config->min_double)
		{
		}
		else {
			temp = make_hipDoubleComplex(0, 0);
		}

		//hipDoubleComplex addtmp = output[col + row * config->pn_X];
		//output[col+row*config->pn_X] = hipCadd(addtmp,temp);

		output[tid].x += temp.x;
		output[tid].y += temp.y;
	}
}

__global__
void cudaKernel_double_RefAS_continuous(hipfftDoubleComplex* output, const MeshKernelConfig* config,
	const geometric* geom, double av0, double av1, double av2, double carrierWaveX, double carrierWaveY, double carrierWaveZ)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if (tid < config->pn_X * config->pn_Y) {

		int col = tid % config->pn_X;
		int row = tid / config->pn_X;

		double flx, fly, flz, fx, fy, fz, flxShifted, flyShifted, freqTermX, freqTermY;

		double det = geom->loRot[0] * geom->loRot[3] - geom->loRot[1] * geom->loRot[2];
		if (det == 0)
			return;

		double a = 1 / det;
		double invLoRot[4];
		invLoRot[0] = a * geom->loRot[3];
		invLoRot[1] = -a * geom->loRot[2];
		invLoRot[2] = -a * geom->loRot[1];
		invLoRot[3] = a * geom->loRot[0];

		hipDoubleComplex refTerm1 = make_hipDoubleComplex(0, 0);
		hipDoubleComplex refTerm2 = make_hipDoubleComplex(0, 0);
		hipDoubleComplex refTerm3 = make_hipDoubleComplex(0, 0);
		hipDoubleComplex refAS = make_hipDoubleComplex(0, 0);
		hipDoubleComplex term1 = make_hipDoubleComplex(0, 0);
		hipDoubleComplex term2 = make_hipDoubleComplex(0, 0);

		term1.y = -config->pi2 / config->lambda * (
			carrierWaveX * (geom->glRot[0] * geom->glShift[0] + geom->glRot[3] * geom->glShift[1] + geom->glRot[6] * geom->glShift[2])
			+ carrierWaveY * (geom->glRot[1] * geom->glShift[0] + geom->glRot[4] * geom->glShift[1] + geom->glRot[7] * geom->glShift[2])
			+ carrierWaveZ * (geom->glRot[2] * geom->glShift[0] + geom->glRot[5] * geom->glShift[1] + geom->glRot[8] * geom->glShift[2]));


		// calculate frequency term =======================================================================
		int idxFx = -config->pn_X / 2 + col;
		int idxFy = config->pn_X / 2 - row;
		double w = 1.0 / config->lambda;

		fx = (double)idxFx * config->dfx;
		fy = (double)idxFy * config->dfy;
		fz = sqrt(w * w - fx * fx - fy * fy);

		flx = geom->glRot[0] * fx + geom->glRot[1] * fy + geom->glRot[2] * fz;
		fly = geom->glRot[3] * fx + geom->glRot[4] * fy + geom->glRot[5] * fz;
		flz = sqrt(w * w - flx * flx - fly * fly);


		flxShifted = flx - w * (geom->glRot[0] * carrierWaveX + geom->glRot[1] * carrierWaveY + geom->glRot[2] * carrierWaveZ);
		flyShifted = fly - w * (geom->glRot[3] * carrierWaveX + geom->glRot[4] * carrierWaveY + geom->glRot[5] * carrierWaveZ);
		freqTermX = invLoRot[0] * flxShifted + invLoRot[1] * flyShifted;
		freqTermY = invLoRot[2] * flxShifted + invLoRot[3] * flyShifted;

		double sqFreqTermX = freqTermX * freqTermX;
		double cuFreqTermX = sqFreqTermX * freqTermX;
		double sqFreqTermY = freqTermY * freqTermY;
		double cuFreqTermY = sqFreqTermY * freqTermY;

		hipDoubleComplex D1 = make_hipDoubleComplex(0, 0);
		hipDoubleComplex D2 = make_hipDoubleComplex(0, 0);
		hipDoubleComplex D3 = make_hipDoubleComplex(0, 0);

		//if (freqTermX == 0.0 && freqTermY == 0.0) {
		if (abs(freqTermX) <= config->tolerence && abs(freqTermY) <= config->tolerence) {

			D1.x = (double)1.0 / (double)3.0;
			D2.x = (double)1.0 / (double)5.0;
			D3.x = (double)1.0 / (double)2.0;

			//}else if (freqTermX == 0.0 && freqTermY != 0.0) {
		}
		else if (abs(freqTermX) <= config->tolerence && abs(freqTermY) > config->tolerence) {

			refTerm1.y = -config->pi2 * freqTermY;
			refTerm2.y = 1;

			//D1 = (refTerm1 - (Real)1)*refTerm1.exp() / (8 * M_PI*M_PI*M_PI*freqTermY * freqTermY * freqTermY)
			//	- refTerm1 / (4 * M_PI*M_PI*M_PI*freqTermY * freqTermY * freqTermY);

			hipDoubleComplex refTerm1_exp = make_hipDoubleComplex(refTerm1.x, refTerm1.y);
			exponent_complex_mesh(&refTerm1_exp);
			hipDoubleComplex value1 = make_hipDoubleComplex(1, 0);
			hipDoubleComplex value2 = hipCsub(refTerm1, value1);
			hipDoubleComplex value3 = hipCmul(value2, refTerm1_exp);
			hipDoubleComplex value4 = hipCdiv(value3, make_hipDoubleComplex(config->cube_pi2 * cuFreqTermY, 0));
			hipDoubleComplex value5 = hipCdiv(refTerm1, make_hipDoubleComplex(config->square_pi2 * config->pi * cuFreqTermY, 0));

			D1 = hipCsub(value4, value5);

			//D2 = -(M_PI*freqTermY + refTerm2) / (4 * M_PI*M_PI*M_PI*freqTermY * freqTermY * freqTermY)*exp(refTerm1)
			//	+ refTerm1 / (8 * M_PI*M_PI*M_PI*freqTermY * freqTermY * freqTermY);
			hipDoubleComplex value6 = hipCadd(make_hipDoubleComplex(config->pi * freqTermY, 0), refTerm2);
			hipDoubleComplex value7 = hipCmul(make_hipDoubleComplex(-1, 0), value6);
			hipDoubleComplex value8 = hipCdiv(value7, make_hipDoubleComplex(config->square_pi2 * config->pi * cuFreqTermY, 0));
			hipDoubleComplex value9 = hipCmul(value8, refTerm1_exp);
			hipDoubleComplex value10 = hipCdiv(refTerm1, make_hipDoubleComplex(config->cube_pi2 * cuFreqTermY, 0));
			D2 = hipCadd(value9, value10);

			//D3 = exp(refTerm1) / (2 * M_PI*freqTermY) + ((Real)1 - refTerm2) / (2 * M_PI*freqTermY);
			hipDoubleComplex value11 = hipCdiv(refTerm1_exp, make_hipDoubleComplex(config->pi2 * freqTermY, 0));
			hipDoubleComplex value12 = hipCsub(make_hipDoubleComplex(1, 0), refTerm2);
			hipDoubleComplex value13 = hipCdiv(value12, make_hipDoubleComplex(config->pi2 * freqTermY, 0));

			D3 = hipCadd(value11, value13);

			//} else if (freqTermX != 0.0 && freqTermY == 0.0) {
		}
		else if (abs(freqTermX) > config->tolerence && abs(freqTermY) <= config->tolerence) {

			refTerm1.y = config->square_pi2 * freqTermX * freqTermX;
			refTerm2.y = 1;
			refTerm3.y = config->pi2 * freqTermX;

			//D1 = (refTerm1 + 4 * M_PI*freqTermX - (Real)2 * refTerm2) / (8 * M_PI*M_PI*M_PI*freqTermY * freqTermY * freqTermY)*exp(-refTerm3)
			//	+ refTerm2 / (4 * M_PI*M_PI*M_PI*freqTermX * freqTermX * freqTermX);

			hipDoubleComplex refTerm3_exp = make_hipDoubleComplex(refTerm3.x, refTerm3.y);
			exponent_complex_mesh(&refTerm3_exp);

			hipDoubleComplex value1 = hipCadd(refTerm1, make_hipDoubleComplex(4 * config->pi * freqTermX, 0));
			hipDoubleComplex value2 = hipCmul(make_hipDoubleComplex(2, 0), refTerm2);
			hipDoubleComplex value3 = hipCsub(value1, value2);
			hipDoubleComplex value4 = hipCdiv(value3, make_hipDoubleComplex(config->cube_pi2 * cuFreqTermY, 0));
			hipDoubleComplex value5 = hipCmul(value4, refTerm3_exp);
			hipDoubleComplex value6 = hipCdiv(refTerm2, make_hipDoubleComplex(config->square_pi2 * config->pi * cuFreqTermX, 0));

			D1 = hipCadd(value5, value6);

			//D2 = (Real)1 / (Real)2 * D1;
			D2 = hipCmul(make_hipDoubleComplex(1.0 / 2.0, 0), D1);

			//D3 = ((refTerm3 + (Real)1)*exp(-refTerm3) - (Real)1) / (4 * M_PI*M_PI*freqTermX * freqTermX);
			hipDoubleComplex value7 = hipCadd(refTerm3, make_hipDoubleComplex(1.0, 0));
			hipDoubleComplex value8 = hipCmul(refTerm3, make_hipDoubleComplex(-1.0, 0));
			exponent_complex_mesh(&value8);
			hipDoubleComplex value9 = hipCmul(value7, value8);
			hipDoubleComplex value10 = hipCsub(value9, make_hipDoubleComplex(1.0, 0));
			D3 = hipCdiv(value10, make_hipDoubleComplex(config->square_pi2 * sqFreqTermX, 0));

			//} else if (freqTermX == -freqTermY) {
		}
		else if (abs(freqTermX + freqTermY) <= config->tolerence) {

			refTerm1.y = 1;
			refTerm2.y = config->pi2 * freqTermX;
			refTerm3.y = config->pi2 * config->pi * freqTermX * freqTermX;

			//D1 = (-2 * M_PI*freqTermX + refTerm1) / (8 * M_PI*M_PI*M_PI*freqTermX * freqTermX * freqTermX)*exp(-refTerm2)
			//	- (refTerm3 + refTerm1) / (8 * M_PI*M_PI*M_PI*freqTermX * freqTermX * freqTermX);

			hipDoubleComplex value1 = hipCadd(make_hipDoubleComplex(-config->pi2 * freqTermX, 0), refTerm1);
			hipDoubleComplex value2 = hipCdiv(value1, make_hipDoubleComplex(config->cube_pi2 * cuFreqTermX, 0));
			hipDoubleComplex value3 = hipCmul(refTerm2, make_hipDoubleComplex(-1.0, 0));
			exponent_complex_mesh(&value3);
			hipDoubleComplex value4 = hipCmul(value2, value3);

			hipDoubleComplex value5 = hipCadd(refTerm3, refTerm1);
			hipDoubleComplex value6 = hipCdiv(value5, make_hipDoubleComplex(config->cube_pi2 * cuFreqTermX, 0));

			D1 = hipCsub(value4, value6);

			//D2 = (-refTerm1) / (8 * M_PI*M_PI*M_PI*freqTermX * freqTermX * freqTermX)*exp(-refTerm2)
			//	+ (-refTerm3 + refTerm1 + 2 * M_PI*freqTermX) / (8 * M_PI*M_PI*M_PI*freqTermX * freqTermX * freqTermX);

			hipDoubleComplex value7 = hipCmul(refTerm1, make_hipDoubleComplex(-1.0, 0));
			hipDoubleComplex value8 = hipCdiv(value7, make_hipDoubleComplex(config->cube_pi2 * cuFreqTermX, 0));
			hipDoubleComplex value9 = hipCmul(value8, value3);

			hipDoubleComplex value10 = hipCmul(refTerm3, make_hipDoubleComplex(-1.0, 0));
			hipDoubleComplex value11 = hipCadd(value10, refTerm1);
			hipDoubleComplex value12 = hipCadd(value11, make_hipDoubleComplex(config->pi2 * freqTermX, 0));
			hipDoubleComplex value13 = hipCdiv(value12, make_hipDoubleComplex(config->cube_pi2 * cuFreqTermX, 0));

			D2 = hipCadd(value9, value13);

			//D3 = (-refTerm1) / (4 * M_PI*M_PI*freqTermX * freqTermX)*exp(-refTerm2)
			//	+ (-refTerm2 + (Real)1) / (4 * M_PI*M_PI*freqTermX * freqTermX);

			hipDoubleComplex value14 = hipCdiv(value7, make_hipDoubleComplex(config->square_pi2 * sqFreqTermX, 0));
			hipDoubleComplex value15 = hipCmul(value14, value3);

			hipDoubleComplex value16 = hipCmul(refTerm2, make_hipDoubleComplex(-1.0, 0));
			hipDoubleComplex value17 = hipCadd(value16, make_hipDoubleComplex(1.0, 0));
			hipDoubleComplex value18 = hipCdiv(value17, make_hipDoubleComplex(config->square_pi2 * sqFreqTermX, 0));

			D3 = hipCadd(value15, value18);

		}
		else {

			refTerm1.y = -config->pi2 * (freqTermX + freqTermY);
			refTerm2.y = 1.0;
			refTerm3.y = -config->pi2 * freqTermX;

			//D1 = exp(refTerm1)*(refTerm2 - 2 * M_PI*(freqTermX + freqTermY)) / (8 * M_PI*M_PI*M_PI*freqTermY * (freqTermX + freqTermY)*(freqTermX + freqTermY))
			//	+ exp(refTerm3)*(2 * M_PI*freqTermX - refTerm2) / (8 * M_PI*M_PI*M_PI*freqTermX * freqTermX * freqTermY)
			//	+ ((2 * freqTermX + freqTermY)*refTerm2) / (8 * M_PI*M_PI*M_PI*freqTermX * freqTermX * (freqTermX + freqTermY)*(freqTermX + freqTermY));

			hipDoubleComplex refTerm1_exp = make_hipDoubleComplex(refTerm1.x, refTerm1.y);
			exponent_complex_mesh(&refTerm1_exp);

			double val1 = config->pi2 * (freqTermX + freqTermY);
			hipDoubleComplex value1 = hipCsub(refTerm2, make_hipDoubleComplex(val1, 0));
			hipDoubleComplex value2 = hipCmul(refTerm1_exp, value1);

			double val2 = config->cube_pi2 * freqTermY * (freqTermX + freqTermY) * (freqTermX + freqTermY);
			hipDoubleComplex value3 = hipCdiv(value2, make_hipDoubleComplex(val2, 0));

			hipDoubleComplex refTerm3_exp = make_hipDoubleComplex(refTerm3.x, refTerm3.y);
			exponent_complex_mesh(&refTerm3_exp);

			double val3 = config->pi2 * freqTermX;
			hipDoubleComplex value4 = hipCsub(make_hipDoubleComplex(val3, 0), refTerm2);
			hipDoubleComplex value5 = hipCmul(refTerm3_exp, value4);
			double val4 = config->cube_pi2 * sqFreqTermX * freqTermY;
			hipDoubleComplex value6 = hipCdiv(value5, make_hipDoubleComplex(val4, 0));

			double val5 = 2.0 * freqTermX + freqTermY;
			hipDoubleComplex value7 = hipCmul(make_hipDoubleComplex(val5, 0), refTerm2);
			double val6 = config->cube_pi2 * sqFreqTermX * (freqTermX + freqTermY) * (freqTermX + freqTermY);
			hipDoubleComplex value8 = hipCdiv(value7, make_hipDoubleComplex(val6, 0));

			hipDoubleComplex value9 = hipCadd(value3, value6);
			D1 = hipCadd(value9, value8);

			//D2 = exp(refTerm1)*(refTerm2*(freqTermX + 2 * freqTermY) - 2 * M_PI*freqTermY * (freqTermX + freqTermY)) / (8 * M_PI*M_PI*M_PI*freqTermY * freqTermY * (freqTermX + freqTermY)*(freqTermX + freqTermY))
			//	+ exp(refTerm3)*(-refTerm2) / (8 * M_PI*M_PI*M_PI*freqTermX * freqTermY * freqTermY)
			//	+ refTerm2 / (8 * M_PI*M_PI*M_PI*freqTermX * (freqTermX + freqTermY)* (freqTermX + freqTermY));

			double val7 = freqTermX + 2.0 * freqTermY;
			hipDoubleComplex value10 = hipCmul(refTerm2, make_hipDoubleComplex(val7, 0));
			double val8 = config->pi2 * freqTermY * (freqTermX + freqTermY);
			hipDoubleComplex value11 = hipCsub(value10, make_hipDoubleComplex(val8, 0));
			hipDoubleComplex value12 = hipCmul(refTerm1_exp, value11);
			double val9 = config->cube_pi2 * sqFreqTermY * (freqTermX + freqTermY) * (freqTermX + freqTermY);
			hipDoubleComplex value13 = hipCdiv(value12, make_hipDoubleComplex(val9, 0));

			hipDoubleComplex value14 = hipCmul(refTerm2, make_hipDoubleComplex(-1.0, 0));
			hipDoubleComplex value15 = hipCmul(refTerm3_exp, value14);
			double val10 = config->cube_pi2 * freqTermX * sqFreqTermY;
			hipDoubleComplex value16 = hipCdiv(value15, make_hipDoubleComplex(val10, 0));

			double val11 = config->cube_pi2 * freqTermX * (freqTermX + freqTermY) * (freqTermX + freqTermY);
			hipDoubleComplex value17 = hipCdiv(refTerm2, make_hipDoubleComplex(val11, 0));

			hipDoubleComplex value18 = hipCadd(value13, value16);
			D2 = hipCadd(value18, value17);

			//D3 = -exp(refTerm1) / (4 * M_PI*M_PI*freqTermY * (freqTermX + freqTermY))
			//	+ exp(refTerm3) / (4 * M_PI*M_PI*freqTermX * freqTermY)
			//	- (Real)1 / (4 * M_PI*M_PI*freqTermX * (freqTermX + freqTermY));

			hipDoubleComplex value19 = hipCmul(refTerm1_exp, make_hipDoubleComplex(-1.0, 0));
			double val12 = config->square_pi2 * freqTermY * (freqTermX + freqTermY);
			hipDoubleComplex value20 = hipCdiv(value19, make_hipDoubleComplex(val12, 0));

			double val13 = config->square_pi2 * freqTermX * freqTermY;
			hipDoubleComplex value21 = hipCdiv(refTerm3_exp, make_hipDoubleComplex(val13, 0));

			double val14 = 1.0 / (config->square_pi2 * freqTermX * (freqTermX + freqTermY));
			hipDoubleComplex value22 = make_hipDoubleComplex(val14, 0);

			hipDoubleComplex value23 = hipCadd(value20, value21);
			D3 = hipCsub(value23, value22);

		}

		//refAS = (av1 - av0)*D1 + (av2 - av1)*D2 + av0 * D3;

		double t1 = av1 - av0;
		double t2 = av2 - av1;
		hipDoubleComplex value_temp1 = hipCmul(make_hipDoubleComplex(t1, 0), D1);
		hipDoubleComplex value_temp2 = hipCmul(make_hipDoubleComplex(t2, 0), D2);
		hipDoubleComplex value_temp3 = hipCmul(make_hipDoubleComplex(av0, 0), D3);

		hipDoubleComplex valeF = hipCadd(value_temp1, value_temp2);
		refAS = hipCadd(valeF, value_temp3);

		hipDoubleComplex temp;
		if (abs(fz) <= config->tolerence)
			temp = make_hipDoubleComplex(0, 0);
		else {
			term2.y = config->pi2 * (flx * geom->glShift[0] + fly * geom->glShift[1] + flz * geom->glShift[2]);

			//temp = refAS / det * exp(term1)* flz / fz * exp(term2);

			exponent_complex_mesh(&term1);
			exponent_complex_mesh(&term2);

			hipDoubleComplex tmp1 = hipCdiv(refAS, make_hipDoubleComplex(det, 0));
			hipDoubleComplex tmp2 = hipCmul(tmp1, term1);
			hipDoubleComplex tmp3 = hipCmul(tmp2, make_hipDoubleComplex(flz, 0));
			hipDoubleComplex tmp4 = hipCdiv(tmp3, make_hipDoubleComplex(fz, 0));
			temp = hipCmul(tmp4, term2);

		}

		double absval = sqrt((temp.x * temp.x) + (temp.y * temp.y));
		if (absval > config->min_double)
		{
		}
		else {
			temp = make_hipDoubleComplex(0, 0);
		}

		//hipDoubleComplex addtmp = output[col + row * config->pn_X];
		//output[col+row*config->pn_X] = hipCadd(addtmp,temp);

		output[tid].x += temp.x;
		output[tid].y += temp.y;
	}
}

extern "C"
{
	void cudaMesh_Flat(
		const int& nBlocks, const int& nThreads, hipfftDoubleComplex* output,
		const MeshKernelConfig* config, double shading_factor, const geometric* geom,
		double carrierWaveX, double carrierWaveY, double carrierWaveZ, ihipStream_t* stream)
	{
		cudaKernel_double_RefAS_flat << <nBlocks, nThreads, 0, stream >> > (output, config, shading_factor,
			geom, carrierWaveX, carrierWaveY, carrierWaveZ);
	}

	void cudaMesh_Continuous(
		const int& nBlocks, const int& nThreads, hipfftDoubleComplex* output,
		const MeshKernelConfig* config, const geometric* geom, double av0, double av1, double av2,
		double carrierWaveX, double carrierWaveY, double carrierWaveZ, ihipStream_t* stream)
	{
		cudaKernel_double_RefAS_continuous << <nBlocks, nThreads, 0, stream >> > (output, config,
			geom, av0, av1, av2, carrierWaveX, carrierWaveY, carrierWaveZ);
	}


	void call_fftGPU(int nx, int ny, hipfftDoubleComplex* input, hipfftDoubleComplex* output, ihipStream_t* streamTriMesh)
	{
		cudaFFT_Mesh(streamTriMesh, nx, ny, input, output, 1);
	}

	void call_fftGPUf(int nx, int ny, hipFloatComplex* input, hipFloatComplex* output, ihipStream_t* streamTriMesh)
	{
		cudaFFT_Meshf(streamTriMesh, nx, ny, input, output, 1);
	}
}

#endif // !ophTriMeshKernel_cu__