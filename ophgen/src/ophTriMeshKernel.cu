#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install, copy or use the software.
//
//
//                           License Agreement
//                For Open Source Digital Holographic Library
//
// Openholo library is free software;
// you can redistribute it and/or modify it under the terms of the BSD 2-Clause license.
//
// Copyright (C) 2017-2024, Korea Electronics Technology Institute. All rights reserved.
// E-mail : contact.openholo@gmail.com
// Web : http://www.openholo.org
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//  1. Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//  2. Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the copyright holder or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
// This software contains opensource software released under GNU Generic Public License,
// NVDIA Software License Agreement, or CUDA supplement to Software License Agreement.
// Check whether software you use contains licensed software.
//
//M*/
#pragma once
#ifndef ophTriMeshKernel_cu__
#define ophTriMeshKernel_cu__
#include "ophKernel.cuh"
#include <hip/hip_runtime_api.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>
#include <vector>

__device__  void exponent_complex_mesh(hipDoubleComplex* val)
{
	double exp_val = exp(val->x);
	double cos_v;
	double sin_v;
	sincos(val->y, &sin_v, &cos_v);

	val->x = exp_val * cos_v;
	val->y = exp_val * sin_v;
}

__device__  void exponent_complex_meshf(hipFloatComplex* val)
{
	float exp_val = expf(val->x);
	float cos_v;
	float sin_v;
	sincosf(val->y, &sin_v, &cos_v);

	val->x = exp_val * cos_v;
	val->y = exp_val * sin_v;
}


void cudaFFT_Mesh(ihipStream_t* stream, int nx, int ny, hipfftDoubleComplex* in_field, hipfftDoubleComplex* output_field, int direction)
{
	unsigned int nblocks = (nx*ny + kBlockThreads - 1) / kBlockThreads;
	int N = nx * ny;
	fftShift << <nblocks, kBlockThreads, 0, stream >> > (N, nx, ny, in_field, output_field, false);

	hipfftHandle plan;

	// fft
	if (hipfftPlan2d(&plan, ny, nx, HIPFFT_Z2Z) != HIPFFT_SUCCESS)
	{
		//LOG("FAIL in creating cufft plan");
		return;
	};

	hipfftResult result;

	if (direction == -1)
		result = hipfftExecZ2Z(plan, output_field, in_field, HIPFFT_FORWARD);
	else
		result = hipfftExecZ2Z(plan, output_field, in_field, HIPFFT_BACKWARD);

	if (result != HIPFFT_SUCCESS)
	{
		//LOG("------------------FAIL: execute cufft, code=%s", result);
		return;
	}

	if (hipDeviceSynchronize() != hipSuccess) {
		//LOG("Cuda error: Failed to synchronize\n");
		return;
	}

	fftShift << < nblocks, kBlockThreads, 0, stream >> > (N, nx, ny, in_field, output_field, false);

	hipfftDestroy(plan);
}

void cudaFFT_Meshf(ihipStream_t* stream, int nx, int ny, hipfftComplex* in_field, hipfftComplex* output_field, int direction)
{
	unsigned int nblocks = (nx*ny + kBlockThreads - 1) / kBlockThreads;
	int N = nx * ny;
	fftShiftf << <nblocks, kBlockThreads, 0, stream >> > (N, nx, ny, in_field, output_field, false);

	hipfftHandle plan;

	// fft
	if (hipfftPlan2d(&plan, ny, nx, HIPFFT_C2C) != HIPFFT_SUCCESS)
	{
		//LOG("FAIL in creating cufft plan");
		return;
	};

	hipfftResult result;

	if (direction == -1)
		result = hipfftExecC2C(plan, output_field, in_field, HIPFFT_FORWARD);
	else
		result = hipfftExecC2C(plan, output_field, in_field, HIPFFT_BACKWARD);

	if (result != HIPFFT_SUCCESS)
	{
		//LOG("------------------FAIL: execute cufft, code=%s", result);
		return;
	}

	if (hipDeviceSynchronize() != hipSuccess) {
		//LOG("Cuda error: Failed to synchronize\n");
		return;
	}

	fftShiftf << < nblocks, kBlockThreads, 0, stream >> > (N, nx, ny, in_field, output_field, false);

	hipfftDestroy(plan);
}

__global__ void cudaKernel_refASf(hipFloatComplex* output, int nx, int ny, float px, float py, unsigned int sflag, int idx, float waveLength,
	float pi, float shadingFactor, float av0, float av1, float av2,
	float glRot0, float glRot1, float glRot2, float glRot3, float glRot4, float glRot5, float glRot6, float glRot7, float glRot8,
	float loRot0, float loRot1, float loRot2, float loRot3, float glShiftX, float glShiftY, float glShiftZ,
	float carrierWaveX, float carrierWaveY, float carrierWaveZ, float min_double, float tolerence)
{
	int tid = threadIdx.x + blockIdx.x*blockDim.x;

	if (tid < nx*ny) {

		int col = tid % nx;
		int row = tid / nx;
		float flx, fly, flz, fx, fy, fz, flxShifted, flyShifted, freqTermX, freqTermY;
		float dfx = (1.0 / px) / (float)nx;
		float dfy = (1.0 / py) / (float)ny;
		float pi2 = pi * 2;
		float sqpi2 = pi2 * pi2;
		float cupi2 = sqpi2 * pi2;


		float det = loRot0 * loRot3 - loRot1 * loRot2;
		if (det == 0)
			return;


		float invLoRot0, invLoRot1, invLoRot2, invLoRot3;
		invLoRot0 = (1 / det)*loRot3;
		invLoRot1 = -(1 / det)*loRot2;
		invLoRot2 = -(1 / det)*loRot1;
		invLoRot3 = (1 / det)*loRot0;
		hipFloatComplex refTerm1 = make_hipFloatComplex(0, 0);
		hipFloatComplex refTerm2 = make_hipFloatComplex(0, 0);
		hipFloatComplex refTerm3 = make_hipFloatComplex(0, 0);
		hipFloatComplex refAS = make_hipFloatComplex(0, 0);
		hipFloatComplex term1 = make_hipFloatComplex(0, 0);
		hipFloatComplex term2 = make_hipFloatComplex(0, 0);

		term1.y = -pi2 / waveLength * (
			carrierWaveX * (glRot0 * glShiftX + glRot3 * glShiftY + glRot6 * glShiftZ)
			+ carrierWaveY * (glRot1 * glShiftX + glRot4 * glShiftY + glRot7 * glShiftZ)
			+ carrierWaveZ * (glRot2 * glShiftX + glRot5 * glShiftY + glRot8 * glShiftZ));

		// calculate frequency term =======================================================================
		int idxFx = -nx / 2 + col;
		int idxFy = nx / 2 - row;
		float w = 1.0 / waveLength;

		fx = (float)idxFx * dfx;
		fy = (float)idxFy * dfy;
		fz = sqrt(w * w - fx * fx - fy * fy);

		flx = glRot0 * fx + glRot1 * fy + glRot2 * fz;
		fly = glRot3 * fx + glRot4 * fy + glRot5 * fz;
		flz = sqrt(w * w - flx * flx - fly * fly);

		flxShifted = flx - w * (glRot0 * carrierWaveX + glRot1 * carrierWaveY + glRot2 * carrierWaveZ);
		flyShifted = fly - w * (glRot3 * carrierWaveX + glRot4 * carrierWaveY + glRot5 * carrierWaveZ);

		freqTermX = invLoRot0 * flxShifted + invLoRot1 * flyShifted;
		freqTermY = invLoRot2 * flxShifted + invLoRot3 * flyShifted;

		float sqFreqTermX = freqTermX * freqTermX;
		float cuFreqTermX = sqFreqTermX * freqTermX;
		float sqFreqTermY = freqTermY * freqTermY;
		float cuFreqTermY = sqFreqTermY * freqTermY;

		//==============================================================================================
		if (sflag == 0) // SHADING_FLAT
		{

			//if (freqTermX == -freqTermY && freqTermY != 0) {
			if (abs(freqTermX - freqTermY) <= tolerence && abs(freqTermY) > tolerence) {
				refTerm1.y = pi2 * freqTermY;
				refTerm2.y = 1;

				//refAS = shadingFactor * (((Complex<Real>)1 - exp(refTerm1)) / (4 * pi*pi*freqTermY * freqTermY) + refTerm2 / (2 * pi*freqTermY));
				exponent_complex_meshf(&refTerm1);
				hipFloatComplex value1 = make_hipFloatComplex(1, 0);
				hipFloatComplex value2 = hipCsubf(value1, refTerm1);
				float value3 = sqpi2 * freqTermY * freqTermY;
				hipFloatComplex value4 = hipCdivf(value2, make_hipFloatComplex(value3, 0));
				hipFloatComplex value5 = hipCdivf(refTerm2, make_hipFloatComplex(pi2 * freqTermY, 0));
				hipFloatComplex value6 = hipCaddf(value4, value5);
				refAS = hipCmulf(value6, make_hipFloatComplex(shadingFactor, 0));

				//}else if (freqTermX == freqTermY && freqTermX == 0) {
			}
			else if (abs(freqTermX - freqTermY) <= tolerence && abs(freqTermX) <= tolerence) {

				//refAS = shadingFactor * 1 / 2;
				refAS = make_hipFloatComplex(shadingFactor*0.5, 0);

				//} else if (freqTermX != 0 && freqTermY == 0) {
			}
			else if (abs(freqTermX) > tolerence && abs(freqTermY) <= tolerence) {

				refTerm1.y = -pi2 * freqTermX;
				refTerm2.y = 1;

				//refAS = shadingFactor * ((exp(refTerm1) - (Complex<Real>)1) / (2 * M_PI*freqTermX * 2 * M_PI*freqTermX) + (refTerm2 * exp(refTerm1)) / (2 * M_PI*freqTermX));
				exponent_complex_meshf(&refTerm1);
				hipFloatComplex value1 = make_hipFloatComplex(1, 0);
				hipFloatComplex value2 = hipCsubf(refTerm1, value1);
				float value3 = sqpi2 * sqFreqTermX;
				hipFloatComplex value4 = hipCdivf(value2, make_hipFloatComplex(value3, 0));

				hipFloatComplex value5 = hipCmulf(refTerm2, refTerm1);
				hipFloatComplex value6 = hipCdivf(value5, make_hipFloatComplex(pi2 * freqTermX, 0));

				hipFloatComplex value7 = hipCaddf(value4, value6);
				refAS = hipCmulf(value7, make_hipFloatComplex(shadingFactor, 0));

				//} else if (freqTermX == 0 && freqTermY != 0) {
			}
			else if (abs(freqTermX) <= tolerence && abs(freqTermY) > tolerence) {

				refTerm1.y = pi2 * freqTermY;
				refTerm2.y = 1;

				//refAS = shadingFactor * (((Complex<Real>)1 - exp(refTerm1)) / (4 * M_PI*M_PI*freqTermY * freqTermY) - refTerm2 / (2 * M_PI*freqTermY));
				exponent_complex_meshf(&refTerm1);
				hipFloatComplex value1 = make_hipFloatComplex(1, 0);
				hipFloatComplex value2 = hipCsubf(value1, refTerm1);
				float value3 = sqpi2 * sqFreqTermY;
				hipFloatComplex value4 = hipCdivf(value2, make_hipFloatComplex(value3, 0));
				hipFloatComplex value5 = hipCdivf(refTerm2, make_hipFloatComplex(pi2 * freqTermY, 0));
				hipFloatComplex value6 = hipCsubf(value4, value5);
				refAS = hipCmulf(value6, make_hipFloatComplex(shadingFactor, 0));

			}
			else {

				refTerm1.y = -pi2 * freqTermX;
				refTerm2.y = -pi2 * (freqTermX + freqTermY);

				//refAS = shadingFactor * ((exp(refTerm1) - (Complex<Real>)1) / (4 * M_PI*M_PI*freqTermX * freqTermY) + ((Complex<Real>)1 - exp(refTerm2)) / (4 * M_PI*M_PI*freqTermY * (freqTermX + freqTermY)));
				exponent_complex_meshf(&refTerm1);
				hipFloatComplex value1 = make_hipFloatComplex(1, 0);
				hipFloatComplex value2 = hipCsubf(refTerm1, value1);
				float value3 = sqpi2 * freqTermX * freqTermY;
				hipFloatComplex value4 = hipCdivf(value2, make_hipFloatComplex(value3, 0));

				exponent_complex_meshf(&refTerm2);
				hipFloatComplex value5 = hipCsubf(make_hipFloatComplex(1, 0), refTerm2);
				float value6 = sqpi2 * freqTermY * (freqTermX + freqTermY);
				hipFloatComplex value7 = hipCdivf(value5, make_hipFloatComplex(value6, 0));

				hipFloatComplex value8 = hipCaddf(value4, value7);
				refAS = hipCmulf(value8, make_hipFloatComplex(shadingFactor, 0));


			}


		}


		else if (sflag == 1) {  // SHADING_CONTINUOUS

			hipFloatComplex D1 = make_hipFloatComplex(0, 0);
			hipFloatComplex D2 = make_hipFloatComplex(0, 0);
			hipFloatComplex D3 = make_hipFloatComplex(0, 0);


			//if (freqTermX == 0.0 && freqTermY == 0.0) {
			if (abs(freqTermX) <= tolerence && abs(freqTermY) <= tolerence) {

				D1.x = (float)1.0 / (float)3.0;
				D2.x = (float)1.0 / (float)5.0;
				D3.x = (float)1.0 / (float)2.0;

				//}else if (freqTermX == 0.0 && freqTermY != 0.0) {
			}
			else if (abs(freqTermX) <= tolerence && abs(freqTermY) > tolerence) {

				refTerm1.y = -pi2 * freqTermY;
				refTerm2.y = 1;

				//D1 = (refTerm1 - (Real)1)*refTerm1.exp() / (8 * M_PI*M_PI*M_PI*freqTermY * freqTermY * freqTermY)
				//	- refTerm1 / (4 * M_PI*M_PI*M_PI*freqTermY * freqTermY * freqTermY);

				hipFloatComplex refTerm1_exp = make_hipFloatComplex(refTerm1.x, refTerm1.y);
				exponent_complex_meshf(&refTerm1_exp);
				hipFloatComplex value1 = make_hipFloatComplex(1, 0);
				hipFloatComplex value2 = hipCsubf(refTerm1, value1);
				hipFloatComplex value3 = hipCmulf(value2, refTerm1_exp);
				hipFloatComplex value4 = hipCdivf(value3, make_hipFloatComplex(cupi2 * cuFreqTermY, 0));
				hipFloatComplex value5 = hipCdivf(refTerm1, make_hipFloatComplex(sqpi2 * pi * cuFreqTermY, 0));

				D1 = hipCsubf(value4, value5);

				//D2 = -(M_PI*freqTermY + refTerm2) / (4 * M_PI*M_PI*M_PI*freqTermY * freqTermY * freqTermY)*exp(refTerm1)
				//	+ refTerm1 / (8 * M_PI*M_PI*M_PI*freqTermY * freqTermY * freqTermY);
				hipFloatComplex value6 = hipCaddf(make_hipFloatComplex(pi * freqTermY, 0), refTerm2);
				hipFloatComplex value7 = hipCmulf(make_hipFloatComplex(-1, 0), value6);
				hipFloatComplex value8 = hipCdivf(value7, make_hipFloatComplex(sqpi2 * pi * cuFreqTermY, 0));
				hipFloatComplex value9 = hipCmulf(value8, refTerm1_exp);
				hipFloatComplex value10 = hipCdivf(refTerm1, make_hipFloatComplex(cupi2 * cuFreqTermY, 0));
				D2 = hipCaddf(value9, value10);

				//D3 = exp(refTerm1) / (2 * M_PI*freqTermY) + ((Real)1 - refTerm2) / (2 * M_PI*freqTermY);
				hipFloatComplex value11 = hipCdivf(refTerm1_exp, make_hipFloatComplex(pi2 * freqTermY, 0));
				hipFloatComplex value12 = hipCsubf(make_hipFloatComplex(1, 0), refTerm2);
				hipFloatComplex value13 = hipCdivf(value12, make_hipFloatComplex(pi2 * freqTermY, 0));

				D3 = hipCaddf(value11, value13);

				//} else if (freqTermX != 0.0 && freqTermY == 0.0) {
			}
			else if (abs(freqTermX) > tolerence && abs(freqTermY) <= tolerence) {

				refTerm1.y = sqpi2 * freqTermX * freqTermX;
				refTerm2.y = 1;
				refTerm3.y = pi2 * freqTermX;

				//D1 = (refTerm1 + 4 * M_PI*freqTermX - (Real)2 * refTerm2) / (8 * M_PI*M_PI*M_PI*freqTermY * freqTermY * freqTermY)*exp(-refTerm3)
				//	+ refTerm2 / (4 * M_PI*M_PI*M_PI*freqTermX * freqTermX * freqTermX);

				hipFloatComplex refTerm3_exp = make_hipFloatComplex(refTerm3.x, refTerm3.y);
				exponent_complex_meshf(&refTerm3_exp);

				hipFloatComplex value1 = hipCaddf(refTerm1, make_hipFloatComplex(4 * pi * freqTermX, 0));
				hipFloatComplex value2 = hipCmulf(make_hipFloatComplex(2, 0), refTerm2);
				hipFloatComplex value3 = hipCsubf(value1, value2);
				hipFloatComplex value4 = hipCdivf(value3, make_hipFloatComplex(cupi2 * cuFreqTermY, 0));
				hipFloatComplex value5 = hipCmulf(value4, refTerm3_exp);
				hipFloatComplex value6 = hipCdivf(refTerm2, make_hipFloatComplex(sqpi2 * pi * cuFreqTermX, 0));

				D1 = hipCaddf(value5, value6);

				//D2 = (Real)1 / (Real)2 * D1;
				D2 = hipCmulf(make_hipFloatComplex(1.0 / 2.0, 0), D1);

				//D3 = ((refTerm3 + (Real)1)*exp(-refTerm3) - (Real)1) / (4 * M_PI*M_PI*freqTermX * freqTermX);
				hipFloatComplex value7 = hipCaddf(refTerm3, make_hipFloatComplex(1.0, 0));
				hipFloatComplex value8 = hipCmulf(refTerm3, make_hipFloatComplex(-1.0, 0));
				exponent_complex_meshf(&value8);
				hipFloatComplex value9 = hipCmulf(value7, value8);
				hipFloatComplex value10 = hipCsubf(value9, make_hipFloatComplex(1.0, 0));
				D3 = hipCdivf(value10, make_hipFloatComplex(sqpi2 * sqFreqTermX, 0));

				//} else if (freqTermX == -freqTermY) {
			}
			else if (abs(freqTermX + freqTermY) <= tolerence) {

				refTerm1.y = 1;
				refTerm2.y = pi2 * freqTermX;
				refTerm3.y = pi2 * pi * freqTermX * freqTermX;

				//D1 = (-2 * M_PI*freqTermX + refTerm1) / (8 * M_PI*M_PI*M_PI*freqTermX * freqTermX * freqTermX)*exp(-refTerm2)
				//	- (refTerm3 + refTerm1) / (8 * M_PI*M_PI*M_PI*freqTermX * freqTermX * freqTermX);

				hipFloatComplex value1 = hipCaddf(make_hipFloatComplex(-pi2 * freqTermX, 0), refTerm1);
				hipFloatComplex value2 = hipCdivf(value1, make_hipFloatComplex(cupi2 * cuFreqTermX, 0));
				hipFloatComplex value3 = hipCmulf(refTerm2, make_hipFloatComplex(-1.0, 0));
				exponent_complex_meshf(&value3);
				hipFloatComplex value4 = hipCmulf(value2, value3);

				hipFloatComplex value5 = hipCaddf(refTerm3, refTerm1);
				hipFloatComplex value6 = hipCdivf(value5, make_hipFloatComplex(cupi2 * cuFreqTermX, 0));

				D1 = hipCsubf(value4, value6);

				//D2 = (-refTerm1) / (8 * M_PI*M_PI*M_PI*freqTermX * freqTermX * freqTermX)*exp(-refTerm2)
				//	+ (-refTerm3 + refTerm1 + 2 * M_PI*freqTermX) / (8 * M_PI*M_PI*M_PI*freqTermX * freqTermX * freqTermX);

				hipFloatComplex value7 = hipCmulf(refTerm1, make_hipFloatComplex(-1.0, 0));
				hipFloatComplex value8 = hipCdivf(value7, make_hipFloatComplex(cupi2 * cuFreqTermX, 0));
				hipFloatComplex value9 = hipCmulf(value8, value3);

				hipFloatComplex value10 = hipCmulf(refTerm3, make_hipFloatComplex(-1.0, 0));
				hipFloatComplex value11 = hipCaddf(value10, refTerm1);
				hipFloatComplex value12 = hipCaddf(value11, make_hipFloatComplex(pi2 * freqTermX, 0));
				hipFloatComplex value13 = hipCdivf(value12, make_hipFloatComplex(cupi2 * cuFreqTermX, 0));

				D2 = hipCaddf(value9, value13);

				//D3 = (-refTerm1) / (4 * M_PI*M_PI*freqTermX * freqTermX)*exp(-refTerm2)
				//	+ (-refTerm2 + (Real)1) / (4 * M_PI*M_PI*freqTermX * freqTermX);

				hipFloatComplex value14 = hipCdivf(value7, make_hipFloatComplex(sqpi2 * sqFreqTermX, 0));
				hipFloatComplex value15 = hipCmulf(value14, value3);

				hipFloatComplex value16 = hipCmulf(refTerm2, make_hipFloatComplex(-1.0, 0));
				hipFloatComplex value17 = hipCaddf(value16, make_hipFloatComplex(1.0, 0));
				hipFloatComplex value18 = hipCdivf(value17, make_hipFloatComplex(sqpi2 * sqFreqTermX, 0));

				D3 = hipCaddf(value15, value18);

			}
			else {

				refTerm1.y = -pi2 * (freqTermX + freqTermY);
				refTerm2.y = 1.0;
				refTerm3.y = -pi2 * freqTermX;

				//D1 = exp(refTerm1)*(refTerm2 - 2 * M_PI*(freqTermX + freqTermY)) / (8 * M_PI*M_PI*M_PI*freqTermY * (freqTermX + freqTermY)*(freqTermX + freqTermY))
				//	+ exp(refTerm3)*(2 * M_PI*freqTermX - refTerm2) / (8 * M_PI*M_PI*M_PI*freqTermX * freqTermX * freqTermY)
				//	+ ((2 * freqTermX + freqTermY)*refTerm2) / (8 * M_PI*M_PI*M_PI*freqTermX * freqTermX * (freqTermX + freqTermY)*(freqTermX + freqTermY));

				hipFloatComplex refTerm1_exp = make_hipFloatComplex(refTerm1.x, refTerm1.y);
				exponent_complex_meshf(&refTerm1_exp);

				float val1 = pi2 * (freqTermX + freqTermY);
				hipFloatComplex value1 = hipCsubf(refTerm2, make_hipFloatComplex(val1, 0));
				hipFloatComplex value2 = hipCmulf(refTerm1_exp, value1);
				float val2 = cupi2 * freqTermY * (freqTermX + freqTermY)*(freqTermX + freqTermY);
				hipFloatComplex value3 = hipCdivf(value2, make_hipFloatComplex(val2, 0));

				hipFloatComplex refTerm3_exp = make_hipFloatComplex(refTerm3.x, refTerm3.y);
				exponent_complex_meshf(&refTerm3_exp);

				float val3 = pi2 * freqTermX;
				hipFloatComplex value4 = hipCsubf(make_hipFloatComplex(val3, 0), refTerm2);
				hipFloatComplex value5 = hipCmulf(refTerm3_exp, value4);
				float val4 = cupi2 * sqFreqTermX * freqTermY;
				hipFloatComplex value6 = hipCdivf(value5, make_hipFloatComplex(val4, 0));

				float val5 = 2.0 * freqTermX + freqTermY;
				hipFloatComplex value7 = hipCmulf(make_hipFloatComplex(val5, 0), refTerm2);
				float val6 = cupi2 * sqFreqTermX * (freqTermX + freqTermY) * (freqTermX + freqTermY);
				hipFloatComplex value8 = hipCdivf(value7, make_hipFloatComplex(val6, 0));

				hipFloatComplex value9 = hipCaddf(value3, value6);
				D1 = hipCaddf(value9, value8);

				//D2 = exp(refTerm1)*(refTerm2*(freqTermX + 2 * freqTermY) - 2 * M_PI*freqTermY * (freqTermX + freqTermY)) / (8 * M_PI*M_PI*M_PI*freqTermY * freqTermY * (freqTermX + freqTermY)*(freqTermX + freqTermY))
				//	+ exp(refTerm3)*(-refTerm2) / (8 * M_PI*M_PI*M_PI*freqTermX * freqTermY * freqTermY)
				//	+ refTerm2 / (8 * M_PI*M_PI*M_PI*freqTermX * (freqTermX + freqTermY)* (freqTermX + freqTermY));

				float val7 = freqTermX + 2.0 * freqTermY;
				hipFloatComplex value10 = hipCmulf(refTerm2, make_hipFloatComplex(val7, 0));
				float val8 = pi2 * freqTermY * (freqTermX + freqTermY);
				hipFloatComplex value11 = hipCsubf(value10, make_hipFloatComplex(val8, 0));
				hipFloatComplex value12 = hipCmulf(refTerm1_exp, value11);
				float val9 = cupi2 * sqFreqTermY * (freqTermX + freqTermY) * (freqTermX + freqTermY);
				hipFloatComplex value13 = hipCdivf(value12, make_hipFloatComplex(val9, 0));

				hipFloatComplex value14 = hipCmulf(refTerm2, make_hipFloatComplex(-1.0, 0));
				hipFloatComplex value15 = hipCmulf(refTerm3_exp, value14);
				float val10 = cupi2 * freqTermX * sqFreqTermY;
				hipFloatComplex value16 = hipCdivf(value15, make_hipFloatComplex(val10, 0));

				float val11 = cupi2 * freqTermX * (freqTermX + freqTermY) * (freqTermX + freqTermY);
				hipFloatComplex value17 = hipCdivf(refTerm2, make_hipFloatComplex(val11, 0));

				hipFloatComplex value18 = hipCaddf(value13, value16);
				D2 = hipCaddf(value18, value17);

				//D3 = -exp(refTerm1) / (4 * M_PI*M_PI*freqTermY * (freqTermX + freqTermY))
				//	+ exp(refTerm3) / (4 * M_PI*M_PI*freqTermX * freqTermY)
				//	- (Real)1 / (4 * M_PI*M_PI*freqTermX * (freqTermX + freqTermY));

				hipFloatComplex value19 = hipCmulf(refTerm1_exp, make_hipFloatComplex(-1.0, 0));
				float val12 = sqpi2 * freqTermY * (freqTermX + freqTermY);
				hipFloatComplex value20 = hipCdivf(value19, make_hipFloatComplex(val12, 0));

				float val13 = sqpi2 * freqTermX * freqTermY;
				hipFloatComplex value21 = hipCdivf(refTerm3_exp, make_hipFloatComplex(val13, 0));

				float val14 = 1.0 / (sqpi2 * freqTermX * (freqTermX + freqTermY));
				hipFloatComplex value22 = make_hipFloatComplex(val14, 0);

				hipFloatComplex value23 = hipCaddf(value20, value21);
				D3 = hipCsubf(value23, value22);

			}

			//refAS = (av1 - av0)*D1 + (av2 - av1)*D2 + av0 * D3;

			float t1 = av1 - av0;
			float t2 = av2 - av1;
			hipFloatComplex value_temp1 = hipCmulf(make_hipFloatComplex(t1, 0), D1);
			hipFloatComplex value_temp2 = hipCmulf(make_hipFloatComplex(t2, 0), D2);
			hipFloatComplex value_temp3 = hipCmulf(make_hipFloatComplex(av0, 0), D3);

			hipFloatComplex valeF = hipCaddf(value_temp1, value_temp2);
			refAS = hipCaddf(valeF, value_temp3);

		}
		hipFloatComplex temp;
		if (abs(fz) <= tolerence)
			temp = make_hipFloatComplex(0, 0);
		else {
			term2.y = pi2 * (flx * glShiftX + fly * glShiftY + flz * glShiftZ);

			//temp = refAS / det * exp(term1)* flz / fz * exp(term2);

			exponent_complex_meshf(&term1);
			exponent_complex_meshf(&term2);

			hipFloatComplex tmp1 = hipCdivf(refAS, make_hipFloatComplex(det, 0));
			hipFloatComplex tmp2 = hipCmulf(tmp1, term1);
			hipFloatComplex tmp3 = hipCmulf(tmp2, make_hipFloatComplex(flz, 0));
			hipFloatComplex tmp4 = hipCdivf(tmp3, make_hipFloatComplex(fz, 0));
			temp = hipCmulf(tmp4, term2);

		}

		float absval = sqrt((temp.x*temp.x) + (temp.y*temp.y));
		if (absval > min_double)
		{
		}
		else {
			temp = make_hipFloatComplex(0, 0);
		}

		//hipFloatComplex addtmp = output[col + row * nx];
		//output[col+row*nx] = hipCaddf(addtmp,temp);

		output[col + row * nx].x = output[col + row * nx].x + temp.x;
		output[col + row * nx].y = output[col + row * nx].y + temp.y;
	}
}


__global__ void cudaKernel_refAS(hipfftDoubleComplex* output, int nx, int ny, double px, double py, unsigned int sflag, int idx, double waveLength, 
	double pi, double shadingFactor, double av0, double av1, double av2,
	double glRot0, double glRot1, double glRot2, double glRot3, double glRot4, double glRot5, double glRot6, double glRot7, double glRot8,
	double loRot0, double loRot1, double loRot2, double loRot3, double glShiftX, double glShiftY, double glShiftZ,
	double carrierWaveX, double carrierWaveY, double carrierWaveZ, double min_double, double tolerence)
{
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	   	
	if (tid < nx*ny) {

		int col = tid % nx;
		int row = tid / nx;
		
		double flx, fly, flz, fx, fy, fz, flxShifted, flyShifted, freqTermX, freqTermY;
		double dfx = (1.0 / px) / (double)nx;
		double dfy = (1.0 / py) / (double)ny;

		double pi2 = pi * 2;
		double sqpi2 = pi2 * pi2;
		double cupi2 = sqpi2 * pi2;

		double det = loRot0 * loRot3 - loRot1 * loRot2;
		if (det == 0)
			return;

		
		double invLoRot0, invLoRot1, invLoRot2, invLoRot3;
		invLoRot0 = (1 / det)*loRot3;
		invLoRot1 = -(1 / det)*loRot2;
		invLoRot2 = -(1 / det)*loRot1;
		invLoRot3 = (1 / det)*loRot0;

		hipDoubleComplex refTerm1 = make_hipDoubleComplex(0, 0);
		hipDoubleComplex refTerm2 = make_hipDoubleComplex(0, 0);
		hipDoubleComplex refTerm3 = make_hipDoubleComplex(0, 0);
		hipDoubleComplex refAS = make_hipDoubleComplex(0, 0);
		hipDoubleComplex term1 = make_hipDoubleComplex(0, 0);
		hipDoubleComplex term2 = make_hipDoubleComplex(0, 0);

		term1.y = -pi2 / waveLength * (
			carrierWaveX * (glRot0 * glShiftX + glRot3 * glShiftY + glRot6 * glShiftZ)
			+ carrierWaveY * (glRot1 * glShiftX + glRot4 * glShiftY + glRot7 * glShiftZ)
			+ carrierWaveZ * (glRot2 * glShiftX + glRot5 * glShiftY + glRot8 * glShiftZ));

		
		// calculate frequency term =======================================================================
		int idxFx = -nx / 2 + col; 
		int idxFy = nx / 2 - row;
		double w = 1.0 / waveLength;

		fx = (double)idxFx * dfx;
		fy = (double)idxFy * dfy;
		fz = sqrt(w * w - fx * fx - fy * fy);

		flx = glRot0 * fx + glRot1 * fy + glRot2 * fz;
		fly = glRot3 * fx + glRot4 * fy + glRot5 * fz;
		flz = sqrt(w * w - flx * flx - fly * fly);


		flxShifted = flx - w * (glRot0 * carrierWaveX + glRot1 * carrierWaveY + glRot2 * carrierWaveZ);
		flyShifted = fly - w * (glRot3 * carrierWaveX + glRot4 * carrierWaveY + glRot5 * carrierWaveZ);
		freqTermX = invLoRot0 * flxShifted + invLoRot1 * flyShifted;
		freqTermY = invLoRot2 * flxShifted + invLoRot3 * flyShifted;
		
		double sqFreqTermX = freqTermX * freqTermX;
		double cuFreqTermX = sqFreqTermX * freqTermX;
		double sqFreqTermY = freqTermY * freqTermY;
		double cuFreqTermY = sqFreqTermY * freqTermY;
		
		//==============================================================================================
		if (sflag == 0) // SHADING_FLAT
		{
			
			//if (freqTermX == -freqTermY && freqTermY != 0) {
			if (abs(freqTermX-freqTermY) <= tolerence && abs(freqTermY) > tolerence) {
				refTerm1.y = pi2 * freqTermY;
				refTerm2.y = 1;

				//refAS = shadingFactor * (((Complex<Real>)1 - exp(refTerm1)) / (4 * pi*pi*freqTermY * freqTermY) + refTerm2 / (2 * pi*freqTermY));
				exponent_complex_mesh(&refTerm1);
				hipDoubleComplex value1 = make_hipDoubleComplex(1, 0);
				hipDoubleComplex value2 = hipCsub(value1, refTerm1);
				double value3 = sqpi2 * freqTermY * freqTermY;
				hipDoubleComplex value4 = hipCdiv(value2, make_hipDoubleComplex(value3, 0));
				hipDoubleComplex value5 = hipCdiv(refTerm2, make_hipDoubleComplex(pi2 * freqTermY, 0));
				hipDoubleComplex value6 = hipCadd(value4, value5);
				refAS = hipCmul(value6, make_hipDoubleComplex(shadingFactor, 0));
			
			//}else if (freqTermX == freqTermY && freqTermX == 0) {
			} else if (abs(freqTermX-freqTermY) <= tolerence && abs(freqTermX) <= tolerence) {

				//refAS = shadingFactor * 1 / 2;
				refAS = make_hipDoubleComplex(shadingFactor*0.5, 0);
			
			//} else if (freqTermX != 0 && freqTermY == 0) {
			} else if (abs(freqTermX) > tolerence && abs(freqTermY) <= tolerence) {

				refTerm1.y = -pi2 * freqTermX;
				refTerm2.y = 1;

				//refAS = shadingFactor * ((exp(refTerm1) - (Complex<Real>)1) / (2 * M_PI*freqTermX * 2 * M_PI*freqTermX) + (refTerm2 * exp(refTerm1)) / (2 * M_PI*freqTermX));
				exponent_complex_mesh(&refTerm1);
				hipDoubleComplex value1 = make_hipDoubleComplex(1, 0);
				hipDoubleComplex value2 = hipCsub(refTerm1, value1);
				double value3 = sqpi2 * sqFreqTermX;
				hipDoubleComplex value4 = hipCdiv(value2, make_hipDoubleComplex(value3, 0));

				hipDoubleComplex value5 = hipCmul(refTerm2, refTerm1);
				hipDoubleComplex value6 = hipCdiv(value5, make_hipDoubleComplex(pi2 * freqTermX, 0));

				hipDoubleComplex value7 = hipCadd(value4, value6);
				refAS = hipCmul(value7, make_hipDoubleComplex(shadingFactor, 0));

			//} else if (freqTermX == 0 && freqTermY != 0) {
			} else if (abs(freqTermX) <= tolerence && abs(freqTermY) > tolerence) {

				refTerm1.y = pi2 * freqTermY;
				refTerm2.y = 1;
				
				//refAS = shadingFactor * (((Complex<Real>)1 - exp(refTerm1)) / (4 * M_PI*M_PI*freqTermY * freqTermY) - refTerm2 / (2 * M_PI*freqTermY));
				exponent_complex_mesh(&refTerm1);
				hipDoubleComplex value1 = make_hipDoubleComplex(1, 0);
				hipDoubleComplex value2 = hipCsub(value1, refTerm1);
				double value3 = sqpi2 * sqFreqTermY;
				hipDoubleComplex value4 = hipCdiv(value2, make_hipDoubleComplex(value3, 0));
				hipDoubleComplex value5 = hipCdiv(refTerm2, make_hipDoubleComplex(pi2 * freqTermY, 0));
				hipDoubleComplex value6 = hipCsub(value4, value5);
				refAS = hipCmul(value6, make_hipDoubleComplex(shadingFactor, 0));
		
			} else {

				refTerm1.y = -pi2 * freqTermX;
				refTerm2.y = -pi2 * (freqTermX + freqTermY);

				//refAS = shadingFactor * ((exp(refTerm1) - (Complex<Real>)1) / (4 * M_PI*M_PI*freqTermX * freqTermY) + ((Complex<Real>)1 - exp(refTerm2)) / (4 * M_PI*M_PI*freqTermY * (freqTermX + freqTermY)));
				exponent_complex_mesh(&refTerm1);
				hipDoubleComplex value1 = make_hipDoubleComplex(1, 0);
				hipDoubleComplex value2 = hipCsub(refTerm1, value1);
				double value3 = sqpi2 * freqTermX * freqTermY;
				hipDoubleComplex value4 = hipCdiv(value2, make_hipDoubleComplex(value3, 0));

				exponent_complex_mesh(&refTerm2);
				hipDoubleComplex value5 = hipCsub(make_hipDoubleComplex(1, 0), refTerm2);
				double value6 = 4 * pi*pi*freqTermY * (freqTermX + freqTermY);
				hipDoubleComplex value7 = hipCdiv(value5, make_hipDoubleComplex(value6, 0));

				hipDoubleComplex value8 = hipCadd(value4, value7);
				refAS = hipCmul(value8, make_hipDoubleComplex(shadingFactor, 0));


			}
			

		} 
		else if (sflag == 1) {  // SHADING_CONTINUOUS
			
			hipDoubleComplex D1 = make_hipDoubleComplex(0, 0);
			hipDoubleComplex D2 = make_hipDoubleComplex(0, 0);
			hipDoubleComplex D3 = make_hipDoubleComplex(0, 0);
			

			//if (freqTermX == 0.0 && freqTermY == 0.0) {
			if (abs(freqTermX) <= tolerence && abs(freqTermY) <= tolerence) {

				D1.x = (double)1.0 / (double)3.0;
				D2.x = (double)1.0 / (double)5.0;
				D3.x = (double)1.0 / (double)2.0;
			
			//}else if (freqTermX == 0.0 && freqTermY != 0.0) {
			}else if (abs(freqTermX) <= tolerence && abs(freqTermY) > tolerence) {

				refTerm1.y = -pi2 * freqTermY;
				refTerm2.y = 1;

				//D1 = (refTerm1 - (Real)1)*refTerm1.exp() / (8 * M_PI*M_PI*M_PI*freqTermY * freqTermY * freqTermY)
				//	- refTerm1 / (4 * M_PI*M_PI*M_PI*freqTermY * freqTermY * freqTermY);
				
				hipDoubleComplex refTerm1_exp = make_hipDoubleComplex(refTerm1.x, refTerm1.y);
				exponent_complex_mesh(&refTerm1_exp);
				hipDoubleComplex value1 = make_hipDoubleComplex(1, 0);
				hipDoubleComplex value2 = hipCsub(refTerm1, value1);
				hipDoubleComplex value3 = hipCmul(value2, refTerm1_exp);
				hipDoubleComplex value4 = hipCdiv(value3, make_hipDoubleComplex(cupi2 * cuFreqTermY, 0));
				hipDoubleComplex value5 = hipCdiv(refTerm1, make_hipDoubleComplex(sqpi2 * pi * cuFreqTermY, 0));

				D1 = hipCsub(value4, value5);
							   
				//D2 = -(M_PI*freqTermY + refTerm2) / (4 * M_PI*M_PI*M_PI*freqTermY * freqTermY * freqTermY)*exp(refTerm1)
				//	+ refTerm1 / (8 * M_PI*M_PI*M_PI*freqTermY * freqTermY * freqTermY);
				hipDoubleComplex value6 = hipCadd(make_hipDoubleComplex(pi * freqTermY, 0), refTerm2);
				hipDoubleComplex value7 = hipCmul(make_hipDoubleComplex(-1, 0), value6);
				hipDoubleComplex value8 = hipCdiv(value7, make_hipDoubleComplex(sqpi2 * pi * cuFreqTermY, 0));
				hipDoubleComplex value9 = hipCmul(value8, refTerm1_exp);
				hipDoubleComplex value10 = hipCdiv(refTerm1, make_hipDoubleComplex(cupi2 * cuFreqTermY, 0));
				D2 = hipCadd(value9, value10);

				//D3 = exp(refTerm1) / (2 * M_PI*freqTermY) + ((Real)1 - refTerm2) / (2 * M_PI*freqTermY);
				hipDoubleComplex value11 = hipCdiv(refTerm1_exp, make_hipDoubleComplex(pi2 * freqTermY, 0));
				hipDoubleComplex value12 = hipCsub(make_hipDoubleComplex(1, 0), refTerm2);
				hipDoubleComplex value13 = hipCdiv(value12, make_hipDoubleComplex(pi2 * freqTermY, 0));

				D3 = hipCadd(value11, value13);
								
			//} else if (freqTermX != 0.0 && freqTermY == 0.0) {
			} else if (abs(freqTermX) > tolerence && abs(freqTermY) <= tolerence) {

				refTerm1.y = sqpi2 * freqTermX * freqTermX;
				refTerm2.y = 1;
				refTerm3.y = pi2 * freqTermX;

				//D1 = (refTerm1 + 4 * M_PI*freqTermX - (Real)2 * refTerm2) / (8 * M_PI*M_PI*M_PI*freqTermY * freqTermY * freqTermY)*exp(-refTerm3)
				//	+ refTerm2 / (4 * M_PI*M_PI*M_PI*freqTermX * freqTermX * freqTermX);

				hipDoubleComplex refTerm3_exp = make_hipDoubleComplex(refTerm3.x, refTerm3.y);
				exponent_complex_mesh(&refTerm3_exp);

				hipDoubleComplex value1 = hipCadd(refTerm1, make_hipDoubleComplex(4 * pi * freqTermX, 0));
				hipDoubleComplex value2 = hipCmul(make_hipDoubleComplex(2, 0), refTerm2);
				hipDoubleComplex value3 = hipCsub(value1, value2);
				hipDoubleComplex value4 = hipCdiv(value3, make_hipDoubleComplex(cupi2 * cuFreqTermY, 0));
				hipDoubleComplex value5 = hipCmul(value4, refTerm3_exp);
				hipDoubleComplex value6 = hipCdiv(refTerm2, make_hipDoubleComplex(sqpi2 * pi * cuFreqTermX, 0));

				D1 = hipCadd(value5, value6);

				//D2 = (Real)1 / (Real)2 * D1;
				D2 = hipCmul(make_hipDoubleComplex(1.0 / 2.0, 0), D1);

				//D3 = ((refTerm3 + (Real)1)*exp(-refTerm3) - (Real)1) / (4 * M_PI*M_PI*freqTermX * freqTermX);
				hipDoubleComplex value7 = hipCadd(refTerm3, make_hipDoubleComplex(1.0, 0));
				hipDoubleComplex value8 = hipCmul(refTerm3, make_hipDoubleComplex(-1.0, 0));
				exponent_complex_mesh(&value8);
				hipDoubleComplex value9 = hipCmul(value7, value8);
				hipDoubleComplex value10 = hipCsub(value9, make_hipDoubleComplex(1.0, 0));
				D3 = hipCdiv(value10, make_hipDoubleComplex(sqpi2 * sqFreqTermX, 0));

			//} else if (freqTermX == -freqTermY) {
			} else if (abs(freqTermX+freqTermY) <= tolerence ) {

				refTerm1.y = 1;
				refTerm2.y = pi2 * freqTermX;
				refTerm3.y = pi2 * pi * freqTermX * freqTermX;

				//D1 = (-2 * M_PI*freqTermX + refTerm1) / (8 * M_PI*M_PI*M_PI*freqTermX * freqTermX * freqTermX)*exp(-refTerm2)
				//	- (refTerm3 + refTerm1) / (8 * M_PI*M_PI*M_PI*freqTermX * freqTermX * freqTermX);

				hipDoubleComplex value1 = hipCadd(make_hipDoubleComplex(-pi2 * freqTermX, 0), refTerm1);
				hipDoubleComplex value2 = hipCdiv(value1, make_hipDoubleComplex(cupi2 * cuFreqTermX, 0));
				hipDoubleComplex value3 = hipCmul(refTerm2, make_hipDoubleComplex(-1.0, 0));
				exponent_complex_mesh(&value3);
				hipDoubleComplex value4 = hipCmul(value2, value3);

				hipDoubleComplex value5 = hipCadd(refTerm3, refTerm1);
				hipDoubleComplex value6 = hipCdiv(value5, make_hipDoubleComplex(cupi2 * cuFreqTermX, 0));

				D1 = hipCsub(value4, value6);

				//D2 = (-refTerm1) / (8 * M_PI*M_PI*M_PI*freqTermX * freqTermX * freqTermX)*exp(-refTerm2)
				//	+ (-refTerm3 + refTerm1 + 2 * M_PI*freqTermX) / (8 * M_PI*M_PI*M_PI*freqTermX * freqTermX * freqTermX);

				hipDoubleComplex value7 = hipCmul(refTerm1, make_hipDoubleComplex(-1.0, 0));
				hipDoubleComplex value8 = hipCdiv(value7, make_hipDoubleComplex(cupi2 * cuFreqTermX, 0));
				hipDoubleComplex value9 = hipCmul(value8, value3);

				hipDoubleComplex value10 = hipCmul(refTerm3, make_hipDoubleComplex(-1.0, 0));
				hipDoubleComplex value11 = hipCadd(value10, refTerm1);
				hipDoubleComplex value12 = hipCadd(value11, make_hipDoubleComplex(pi2 * freqTermX, 0));
				hipDoubleComplex value13 = hipCdiv(value12, make_hipDoubleComplex(cupi2 * cuFreqTermX, 0));

				D2 = hipCadd(value9, value13);

				//D3 = (-refTerm1) / (4 * M_PI*M_PI*freqTermX * freqTermX)*exp(-refTerm2)
				//	+ (-refTerm2 + (Real)1) / (4 * M_PI*M_PI*freqTermX * freqTermX);

				hipDoubleComplex value14 = hipCdiv(value7, make_hipDoubleComplex(sqpi2 * sqFreqTermX, 0));
				hipDoubleComplex value15 = hipCmul(value14, value3);

				hipDoubleComplex value16 = hipCmul(refTerm2, make_hipDoubleComplex(-1.0, 0));
				hipDoubleComplex value17 = hipCadd(value16, make_hipDoubleComplex(1.0, 0));
				hipDoubleComplex value18 = hipCdiv(value17, make_hipDoubleComplex(sqpi2 * sqFreqTermX, 0));

				D3 = hipCadd(value15, value18);

			} else {

				refTerm1.y = -pi2 * (freqTermX + freqTermY);
				refTerm2.y = 1.0;
				refTerm3.y = -pi2 * freqTermX;

				//D1 = exp(refTerm1)*(refTerm2 - 2 * M_PI*(freqTermX + freqTermY)) / (8 * M_PI*M_PI*M_PI*freqTermY * (freqTermX + freqTermY)*(freqTermX + freqTermY))
				//	+ exp(refTerm3)*(2 * M_PI*freqTermX - refTerm2) / (8 * M_PI*M_PI*M_PI*freqTermX * freqTermX * freqTermY)
				//	+ ((2 * freqTermX + freqTermY)*refTerm2) / (8 * M_PI*M_PI*M_PI*freqTermX * freqTermX * (freqTermX + freqTermY)*(freqTermX + freqTermY));

				hipDoubleComplex refTerm1_exp = make_hipDoubleComplex(refTerm1.x, refTerm1.y);
				exponent_complex_mesh(&refTerm1_exp);

				double val1 = pi2 * (freqTermX + freqTermY);
				hipDoubleComplex value1 = hipCsub(refTerm2, make_hipDoubleComplex(val1, 0));
				hipDoubleComplex value2 = hipCmul(refTerm1_exp, value1);

				double val2 = cupi2 * freqTermY * (freqTermX + freqTermY)*(freqTermX + freqTermY); 
				hipDoubleComplex value3 = hipCdiv(value2, make_hipDoubleComplex(val2,0));

				hipDoubleComplex refTerm3_exp = make_hipDoubleComplex(refTerm3.x, refTerm3.y);
				exponent_complex_mesh(&refTerm3_exp);

				double val3 = pi2 * freqTermX;
				hipDoubleComplex value4 = hipCsub(make_hipDoubleComplex(val3, 0), refTerm2);
				hipDoubleComplex value5 = hipCmul(refTerm3_exp, value4);
				double val4 = cupi2 * sqFreqTermX * freqTermY;
				hipDoubleComplex value6 = hipCdiv(value5, make_hipDoubleComplex(val4,0));

				double val5 = 2.0 * freqTermX + freqTermY;
				hipDoubleComplex value7 = hipCmul(make_hipDoubleComplex(val5,0), refTerm2);
				double val6 = cupi2 * sqFreqTermX * (freqTermX + freqTermY) * (freqTermX + freqTermY);
				hipDoubleComplex value8 = hipCdiv(value7, make_hipDoubleComplex(val6,0));

				hipDoubleComplex value9 = hipCadd(value3, value6);
				D1 = hipCadd(value9, value8);

				//D2 = exp(refTerm1)*(refTerm2*(freqTermX + 2 * freqTermY) - 2 * M_PI*freqTermY * (freqTermX + freqTermY)) / (8 * M_PI*M_PI*M_PI*freqTermY * freqTermY * (freqTermX + freqTermY)*(freqTermX + freqTermY))
				//	+ exp(refTerm3)*(-refTerm2) / (8 * M_PI*M_PI*M_PI*freqTermX * freqTermY * freqTermY)
				//	+ refTerm2 / (8 * M_PI*M_PI*M_PI*freqTermX * (freqTermX + freqTermY)* (freqTermX + freqTermY));
							   
				double val7 = freqTermX + 2.0 * freqTermY;
				hipDoubleComplex value10 = hipCmul(refTerm2, make_hipDoubleComplex(val7,0));
				double val8 = pi2 * freqTermY * (freqTermX + freqTermY);
				hipDoubleComplex value11 = hipCsub(value10, make_hipDoubleComplex(val8,0));
				hipDoubleComplex value12 = hipCmul(refTerm1_exp, value11);
				double val9 = cupi2 * sqFreqTermY * (freqTermX + freqTermY) * (freqTermX + freqTermY);
				hipDoubleComplex value13 = hipCdiv(value12, make_hipDoubleComplex(val9,0));
				
				hipDoubleComplex value14 = hipCmul(refTerm2, make_hipDoubleComplex(-1.0, 0));
				hipDoubleComplex value15 = hipCmul(refTerm3_exp, value14);
				double val10 = cupi2 * freqTermX * sqFreqTermY;
				hipDoubleComplex value16 = hipCdiv(value15, make_hipDoubleComplex(val10,0));

				double val11 = cupi2 * freqTermX * (freqTermX + freqTermY) * (freqTermX + freqTermY);
				hipDoubleComplex value17 = hipCdiv(refTerm2, make_hipDoubleComplex(val11,0));

				hipDoubleComplex value18 = hipCadd(value13, value16);
				D2 = hipCadd(value18, value17);
				
				//D3 = -exp(refTerm1) / (4 * M_PI*M_PI*freqTermY * (freqTermX + freqTermY))
				//	+ exp(refTerm3) / (4 * M_PI*M_PI*freqTermX * freqTermY)
				//	- (Real)1 / (4 * M_PI*M_PI*freqTermX * (freqTermX + freqTermY));

				hipDoubleComplex value19 = hipCmul(refTerm1_exp, make_hipDoubleComplex(-1.0, 0));
				double val12 = sqpi2 * freqTermY * (freqTermX + freqTermY);
				hipDoubleComplex value20 = hipCdiv(value19, make_hipDoubleComplex(val12,0));

				double val13 = sqpi2 * freqTermX * freqTermY;
				hipDoubleComplex value21 = hipCdiv(refTerm3_exp, make_hipDoubleComplex(val13,0));

				double val14 = 1.0 / (sqpi2 * freqTermX * (freqTermX + freqTermY));
				hipDoubleComplex value22 = make_hipDoubleComplex(val14,0);

				hipDoubleComplex value23 = hipCadd(value20, value21);
				D3 = hipCsub(value23, value22);

			}

			//refAS = (av1 - av0)*D1 + (av2 - av1)*D2 + av0 * D3;
	
			double t1 = av1 - av0;
			double t2 = av2 - av1;
			hipDoubleComplex value_temp1 = hipCmul(make_hipDoubleComplex(t1, 0), D1);
			hipDoubleComplex value_temp2 = hipCmul(make_hipDoubleComplex(t2, 0), D2);
			hipDoubleComplex value_temp3 = hipCmul(make_hipDoubleComplex(av0, 0), D3);

			hipDoubleComplex valeF = hipCadd(value_temp1, value_temp2);
			refAS = hipCadd(valeF, value_temp3);
	
		}
			
		hipDoubleComplex temp;
		if (abs(fz) <= tolerence)
			temp = make_hipDoubleComplex(0, 0);
		else {
			term2.y = pi2 * (flx * glShiftX + fly * glShiftY + flz * glShiftZ);

			//temp = refAS / det * exp(term1)* flz / fz * exp(term2);

			exponent_complex_mesh(&term1);
			exponent_complex_mesh(&term2);

			hipDoubleComplex tmp1 = hipCdiv(refAS, make_hipDoubleComplex(det,0));
			hipDoubleComplex tmp2 = hipCmul(tmp1, term1);
			hipDoubleComplex tmp3 = hipCmul(tmp2, make_hipDoubleComplex(flz, 0));
			hipDoubleComplex tmp4 = hipCdiv(tmp3, make_hipDoubleComplex(fz, 0));
			temp = hipCmul(tmp4, term2);
			
		}

		double absval = sqrt((temp.x*temp.x) + (temp.y*temp.y));
		if (absval > min_double)
		{
		} else { 
			temp = make_hipDoubleComplex(0, 0); 
		}

		//hipDoubleComplex addtmp = output[col + row * nx];
		//output[col+row*nx] = hipCadd(addtmp,temp);

		output[col + row * nx].x = output[col + row * nx].x + temp.x;
		output[col + row * nx].y = output[col + row * nx].y + temp.y;
		
	}

}

extern "C"
void call_cudaKernel_refAS(hipfftDoubleComplex* output, int nx, int ny, double px, double py, unsigned int sflag, int idx, double waveLength, 
	double pi, double shadingFactor, double av0, double av1, double av2,
	double glRot0, double glRot1, double glRot2, double glRot3, double glRot4, double glRot5, double glRot6, double glRot7, double glRot8,
	double loRot0, double loRot1, double loRot2, double loRot3, double glShiftX, double glShiftY, double glShiftZ,
	double carrierWaveX, double carrierWaveY, double carrierWaveZ, double min_double, double tolerence, ihipStream_t* streamTriMesh)
{
	dim3 grid((nx*ny + kBlockThreads - 1) / kBlockThreads, 1, 1);
	cudaKernel_refAS << <grid, kBlockThreads, 0, streamTriMesh >> > (output, nx, ny, px, py, sflag, idx, waveLength, pi, shadingFactor, av0, av1, av2,
		glRot0, glRot1, glRot2, glRot3, glRot4, glRot5, glRot6, glRot7, glRot8,
		loRot0, loRot1, loRot2, loRot3, glShiftX, glShiftY, glShiftZ,
		carrierWaveX, carrierWaveY, carrierWaveZ, min_double, tolerence);	   	 
}

extern "C"
void call_cudaKernel_refASf(hipFloatComplex* output, int nx, int ny, float px, float py, unsigned int sflag, int idx, float waveLength,
	float pi, float shadingFactor, float av0, float av1, float av2,
	float glRot0, float glRot1, float glRot2, float glRot3, float glRot4, float glRot5, float glRot6, float glRot7, float glRot8,
	float loRot0, float loRot1, float loRot2, float loRot3, float glShiftX, float glShiftY, float glShiftZ,
	float carrierWaveX, float carrierWaveY, float carrierWaveZ, float min_double, float tolerence, ihipStream_t* streamTriMesh)
{
	dim3 grid((nx*ny + kBlockThreads - 1) / kBlockThreads, 1, 1);
	cudaKernel_refASf << <grid, kBlockThreads, 0, streamTriMesh >> > (output, nx, ny, px, py, sflag, idx, waveLength, pi, shadingFactor, av0, av1, av2,
		glRot0, glRot1, glRot2, glRot3, glRot4, glRot5, glRot6, glRot7, glRot8,
		loRot0, loRot1, loRot2, loRot3, glShiftX, glShiftY, glShiftZ,
		carrierWaveX, carrierWaveY, carrierWaveZ, min_double, tolerence);
}

extern "C"
void call_fftGPU(int nx, int ny, hipfftDoubleComplex* input, hipfftDoubleComplex* output, ihipStream_t* streamTriMesh)
{	
	cudaFFT_Mesh(streamTriMesh, nx, ny, input, output, 1);	   
}

extern "C"
void call_fftGPUf(int nx, int ny, hipFloatComplex* input, hipFloatComplex* output, ihipStream_t* streamTriMesh)
{
	cudaFFT_Meshf(streamTriMesh, nx, ny, input, output, 1);
}


#endif // !ophTriMeshKernel_cu__